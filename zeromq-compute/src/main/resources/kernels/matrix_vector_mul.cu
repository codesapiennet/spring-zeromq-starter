
#include <hip/hip_runtime.h>
// matrix_vector_mul.cu - CUDA kernel for matrix-vector multiplication
// Placeholder kernel. For production use cuBLAS or highly-optimized bespoke kernels.

extern "C" __global__ void matVecMul(const float* matrix, const float* vector, float* result, int rows, int cols) {
    int r = blockIdx.x * blockDim.x + threadIdx.x;
    if (r >= rows) return;

    float sum = 0.0f;
    int base = r * cols;
    for (int c = 0; c < cols; ++c) {
        sum += matrix[base + c] * vector[c];
    }
    result[r] = sum;
} 